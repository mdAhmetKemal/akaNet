#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "ConnectCuda.h"
#include <hip/hip_runtime.h>




__global__ void accuracyMultinominalCompareCu(const  float* __restrict__  outLayer,
	const  int* __restrict__  whichBigIndex,
	float * ErrorArray, int sampleTotal, int sizePiece, int totalPikselSize, int step){
int samplNumIn = blockIdx.x*blockDim.x + threadIdx.x;

if (samplNumIn < sizePiece){
	if ( outLayer[whichBigIndex[samplNumIn]] == 1.0){
		atomicAdd(&ErrorArray[step], 1.0 / sampleTotal);
	//	printf("\n%d    target: %.5f ", whichBigIndex[samplNumIn], outLayer[whichBigIndex[samplNumIn]]);
	}
}
}

__global__ void accuracyMultinominalCu(const  float* __restrict__  targetLayer,
	const  float* __restrict__ outLayer, float * ErrorArray,
	int totalPikselSize, int sizePiece, int sampleTotal,int step){
	int samplNumIn = blockIdx.x*blockDim.x + threadIdx.x;
	int bigIndex = 0;
	float bigValue = 0.0;
	if (samplNumIn < sizePiece){
		
		for (int p = 0; p<totalPikselSize; p++){
		//	printf("\nbigValue %.4f  outlayer%.4f  index%d", bigValue, outLayer[samplNumIn*totalPikselSize + p], p);

			if (outLayer[samplNumIn*totalPikselSize + p] > bigValue){
				bigValue = outLayer[samplNumIn*totalPikselSize + p];
				bigIndex = p;
				
			}
		}
		// printf("\n--bigValue %.4f  outlayer%.4f  index%d", targetLayer[samplNumIn*totalPikselSize + bigIndex],
			// outLayer[samplNumIn*totalPikselSize + bigIndex], bigIndex);
		if (targetLayer[bigIndex] == 1.0){
			atomicAdd(&ErrorArray[step], 1.0 / sampleTotal);
		}
	}
}
void accuracyMultinominal(float* OutLayer, float * targetLayer, float* errorArray, int sizePiksel,
	int sampleTotal, int sizePiece, int epoch, int widthScreen){

	if (epoch%widthScreen == 0){
		hipMemset(errorArray, 0, widthScreen* sizeof(float));
	}
	int totalPikselSize = sizePiksel / sizePiece;
	int block = 8;
	dim3 BlockSize(block);
	dim3 GridSize((sizePiece + block - 1) / BlockSize.x);
	accuracyMultinominalCu << <GridSize, BlockSize >> >(OutLayer, targetLayer,errorArray, totalPikselSize,
		sizePiece, sampleTotal, (epoch%widthScreen));

};




