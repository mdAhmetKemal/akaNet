#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "ConnectCuda.h"
#include <hip/hip_runtime.h>



__device__ inline float logError(float error){
	if (error > 1){
		error =(error/10)+200;
	}
	else if (error < 1){
		error *= 200;
	}
	return  error ;
}

__global__ void weigthShower(int boyut, uchar4 * other_out, int W, int H,
	const  float* __restrict__ WeigthData, int sizeWeigth) {

	const int c = blockIdx.x*blockDim.x + threadIdx.x;
	const int r = blockIdx.y*blockDim.y + threadIdx.y;

	if ((c < W) && (r < H) && (r < boyut) && (c < boyut*sizeWeigth)) {

		int i = (c)+(r)*W; 
		int we = int(r/boyut);
		int he = int(c/ boyut);
		
		if (he <= sizeWeigth && we == 0){
			float value = WeigthData[he];
			if (value < 0){
				value *= -1;
				other_out[i].x = 0;
				other_out[i].y = 0;
				other_out[i].z = unsigned(fminf(254.,(logError( value)))) ;
				other_out[i].w = 255;
				//printf(" \n c: %d,    ", other_out[i].z );
			}
			else if (value == 0){
				other_out[i].x = 127;
				other_out[i].y = 127;
				other_out[i].z = 127;
				other_out[i].w = 255;
			}
			else if (value > 0){
				other_out[i].x = unsigned(fminf(254., (logError( value))));
				other_out[i].y = 0;
				other_out[i].z = 0;
				other_out[i].w = 255;
			}
		}
	
	}
}


__global__ void weigthActivShower(int boyut, uchar4 * other_out, int W, int H,
	const  float* __restrict__ WeigthData, int sizeWeigth, const  float* __restrict__ ActData, 
	int actSize) {

	const int c = blockIdx.x*blockDim.x + threadIdx.x;
	const int r = blockIdx.y*blockDim.y + threadIdx.y;

	if ((c < W) && (r < H) && (r < boyut*(actSize+1)) && (c < boyut*actSize)) {
		int square = actSize;
		int i = (c)+(r)*W;
		int we = int(r / boyut);
		int he = int(c / boyut);

		if (he <= sizeWeigth && we ==1){
			float value = WeigthData[he];
			if (value < 0){
				value *= -1;
				other_out[i].x = 255 - unsigned(fminf(254., (logError(value))));
				other_out[i].y = 255 - unsigned(fminf(254., (logError(value))));
				other_out[i].z = 255;
				other_out[i].w = 255;
				//printf(" \n c: %d,    ", other_out[i].z );
			}
			else if (value == 0){
				other_out[i].x = 127;
				other_out[i].y = 127;
				other_out[i].z = 127;
				other_out[i].w = 255;
			}
			else if (value > 0){
				other_out[i].x = 255;
				other_out[i].y = 255 - unsigned(fminf(254., (logError(value))));
				other_out[i].z = 255 - unsigned(fminf(254., (logError(value))));
				other_out[i].w = 255;
			}
		}
		i = (c)+(r+boyut*3)*W;
		we = int(r / boyut);
		he = int(c / boyut);

		if (he < square && we < square){
			float value = ActData[he + square*we];
			if (value < 0){
				value *= -1;
				other_out[i].x = 255 - unsigned(fminf(254., (logError(value))));
				other_out[i].y = 255 - unsigned(fminf(254., (logError(value))));
				other_out[i].z = 255;
				other_out[i].w = 255;
				//printf(" \n c: %d,    ", other_out[i].z );
			}
			else if (value > 0){
				other_out[i].x = 255;
				other_out[i].y = 255 - unsigned(fminf(254., (logError(value))));
				other_out[i].z = 255 - unsigned(fminf(254., (logError(value))));
				other_out[i].w = 255;
			}else 
			{
				other_out[i].x = 127;
				other_out[i].y = 127;
				other_out[i].z = 127;
				other_out[i].w = 255;
			}
			
		}

	}
}





extern void preComWeigthShower(int boyut, uchar4 * other_out, int W, int H, float * WeigthData,
	int sizeWeigth, float *ActivData, int actSize){
	if (actSize == 0){
		int block = 32;
		dim3 blockSize(block, block);
		dim3 gridSize((W + block - 1) / block, (H + block - 1) / block);
		weigthShower << <gridSize, blockSize >> >(boyut,other_out, W, H,WeigthData,sizeWeigth);
	}
	else{
		actSize = sqrt(actSize);
		int block = 32;
		dim3 blockSize(block, block);
		dim3 gridSize((W + block - 1) / block, (H + block - 1) / block);
		weigthActivShower << <gridSize, blockSize >> >(boyut, other_out, W, H, WeigthData, sizeWeigth,ActivData,actSize);
	}
}