#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "ShowNetCuda.h"

__global__ void connectShowerCuda(uchar4 * other_out, int pikSize, int W, int H,int areaW,int areaH,
	const float * __restrict__ kernel, const float * __restrict__ paralelKernel,
	 int kernelFeature, int kernelHeWi, int kernelChanal,int maxSquare, int maxFeatureBorder){
	const int scrW = blockIdx.x*blockDim.x + threadIdx.x;
	const int scrH = blockIdx.y*blockDim.y + threadIdx.y;
	const int cha = blockIdx.z*blockDim.z + threadIdx.z;
	int  thisFeature, thisHeigth, thisWeigth;

	if (scrW <areaW && scrH < areaH){
		int outIndex = H*(scrH+2*pikSize)+(W-areaW+scrW);
		int thisFeature = maxSquare * int(scrH / (pikSize*(kernelHeWi + 1))) + int(scrW / (pikSize*(kernelHeWi + 1)));
	
			thisHeigth = (scrH % (pikSize*(kernelHeWi + 1))) / pikSize;
			thisWeigth = (scrW % (pikSize*(kernelHeWi + 1))) / pikSize;
			if (false){
				printf("\n  keF:%d kethF:%d maxFe:%d kH:%d thiH:%d kerW:%d thiW:%d H:%d W:%d piSi:%d ",
					kernelFeature, thisFeature, maxSquare, kernelHeWi, thisHeigth, kernelHeWi,
					thisWeigth, scrH, scrW, areaW);
			}
			if (thisFeature < kernelFeature){
				if (thisHeigth == kernelHeWi || thisWeigth == kernelHeWi){
					other_out[outIndex].x = unsigned(240);
					other_out[outIndex].y = unsigned(100);
					other_out[outIndex].z = other_out[outIndex].x;
					other_out[outIndex].w = 255;
				}
				else {
					//int inIndex = (kernelWeigth *(kernelHeigth *(kernelSample*(0) + thisSample) + thisHeigth) + thisWeigth);
					int inIndex = kernelFeature*(kernelHeWi*(kernelHeWi*(cha)+thisWeigth) + thisHeigth) + thisFeature;
					other_out[outIndex].x = unsigned(kernel[inIndex] *25500);
					other_out[outIndex].y = unsigned(paralelKernel[inIndex] *1280000);
					other_out[outIndex].z = 0;
					other_out[outIndex].w = 255;
					//printf("\n %.12f", kernel[inIndex]);
					//*******sorunlar� burdaki g�stergecler ile bulabilirsin 
				}
			}

			else{
				other_out[outIndex].x = unsigned(120);
				other_out[outIndex].y = unsigned(240);
				other_out[outIndex].z = other_out[outIndex].x;
				other_out[outIndex].w = 255;
			}

	}
}

extern void ConnectShower(int pikSize, uchar4 * other_out, int W, int H, Connect * showingConnect){

	float * dataPtr = showingConnect->Tensor;
	float * dataParalelPtr = showingConnect->paralelTensor;
	int kernelFeature = showingConnect->hTfeatureNum;
	int kernelChanal = showingConnect->hTchanal;
	int kernelHeWi = showingConnect->hTheigth;


	int maxFeatureBorder = (H / 2) / ((kernelHeWi + 1)*pikSize);
	int maxSquare = 1;
	while (maxSquare*maxSquare <= kernelFeature &&   maxSquare <= maxFeatureBorder){
		maxSquare++;
	}
	int areaW = maxSquare* ((kernelHeWi + 1)*pikSize);
	int areaH = maxSquare* ((kernelHeWi + 1)*pikSize);
	if (kernelChanal == 1){

		int block = 8;
		dim3 blockSize(block, block, 1);
		dim3 gridSize((areaW + blockSize.x - 1) / blockSize.x, (areaH + blockSize.y - 1) / blockSize.y, kernelChanal);

		connectShowerCuda << <gridSize, blockSize >> > (other_out, pikSize, W, H, areaW, areaH, dataPtr, dataParalelPtr,
			kernelFeature, kernelHeWi, kernelChanal, maxSquare, maxFeatureBorder);
	}


}