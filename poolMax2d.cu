#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "ConnectCuda.h"

__global__ void maxPoolCu(const  float* __restrict__ inputData, float * outData, int sample, int feature,
	int chanal, int outHe, int outWi, int poolScale){
	int forHe = blockIdx.x*blockDim.x + threadIdx.x;
	int forWi = blockIdx.y*blockDim.y + threadIdx.y;
	float maxPiks;
	if (forHe < outHe &&forWi < outWi){
		for (int forSam = 0; forSam < sample; forSam++){
			for (int forFea = 0; forFea < feature; forFea++){
				for (int forCha = 0; forCha < chanal; forCha++){
					int outIndex =sample*(feature*(outHe*(outWi*(forCha)+forWi) + forHe) + forFea) + forSam;
					maxPiks = 0.0;
					for (int poolX = 0; poolX < poolScale; poolX++){
						for (int poolY = 0; poolY < poolScale; poolY++){
							int inIndex=sample*(feature*(outHe*(outWi*(forCha)+(forWi*poolScale)+poolY) + (forHe*poolScale)+poolX) + forFea) + forSam;
						//	printf("\n %.6f ", inputData[inIndex]);
							if (inputData[inIndex] > maxPiks)
								maxPiks = inputData[inIndex];

						}
					}
					outData[outIndex] = maxPiks*100;
					//printf("\n-%.6f ", outData[outIndex]);
				}
			}
		}
	}

}

void poolMax2d(Layer * inputLayer, Layer* outLayer, int poolScale){
	float * inputData = inputLayer->Tensor;
	float * outputData = outLayer->Tensor;
	int inSam = inputLayer->hTsample;
	int inFea = inputLayer->hTfeatureNum;
	int inHe = inputLayer->hTheigth;
	int inWi = inputLayer->hTwidth;
	int inCha = inputLayer->hTchanal;
	int outSam = outLayer->hTsample;
	int outFea = outLayer->hTfeatureNum;
	int outHe = outLayer->hTheigth;
	int outWi = outLayer->hTwidth;
	int outCha = outLayer->hTchanal;
	if (inSam == outSam && inFea == outFea  &&  inCha == outCha){
		if (inHe == outHe*poolScale  &&  inWi == outWi*poolScale){
			dim3 bloksize(16, 16);
			dim3 gridsize((outHe + bloksize.x - 1) / bloksize.x, (outWi + bloksize.y - 1) / bloksize.y);
			maxPoolCu << <gridsize, bloksize >> >(inputData, outputData, inSam, inFea, inCha, outHe, outWi, poolScale);
		}
		else printf("\n PoolLayer Scale Problem");
	}
	else printf("\n PoolLayer Scale Problem");
}