#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "ConnectCuda.h"
#include <hip/hip_runtime.h>

__device__ inline float logError(float error){
	return  error=  (__log10f(error));//15
}
__global__ void error2TextureDevice(const  float* __restrict__ accuracyTrain,
	const  float* __restrict__ accuracyTest,
	const  float* __restrict__  errorArray,
	uchar4 *d_out, int w, int h) {
	const int c = blockIdx.x*blockDim.x + threadIdx.x;
	const int r = blockIdx.y*blockDim.y + threadIdx.y;
	float WE = float(w), HE = float(h), Re = float(r),
		Co = float(c);
	float  pixAcc;
	int basamak = 20;//20
	
	if ((c < w) && (r < h)) {
		//printf("\nxxxx:  %.10f", errorArray[c]);
		int i = (c)+(h - r)*w; // 1D indexing
		//printf("  c: %d,   r: %d     i: %d  ", c, r,i);
		d_out[i].x = 255;
		d_out[i].y = 255;
		d_out[i].z = 255;
		d_out[i].w = 255;
		if ((accuracyTrain[c] / 2) >= (Re / HE)){

				d_out[i].x = 60;
				d_out[i].y = 170;
				d_out[i].z = 80;
				}
		/*else if (r < w / 2 && (accuracyTrain[c] / 2) != 0){
				d_out[i].x = 180;
				d_out[i].y = 200;
				d_out[i].z = 210;
			} */

			if ((accuracyTest[c ] / 2) >= (Re / HE)){
				d_out[i].x += 60;
				d_out[i].y -= 100;
				d_out[i].z += 45;
			} 
				
	/*	pixAcc = logError(errorArray[c]) / (basamak/2);
		//pixAcc = logError(10.0) / (basamak);  //****error 1. nereye denk
		if (pixAcc >= (Re / HE)){
			d_out[i].x +=40;
			d_out[i].y -= 70;
		} */
		
		if (r % (w / (basamak * 5)) == 0){
			d_out[i].y -= 10;
			if (r % ((w / basamak)) == 0){
				d_out[i].x -= 15;
				d_out[i].y -= 15;
				if (r % (int(w / (basamak / 5))) == 0){
					d_out[i].x -= 25;
					d_out[i].y -= 30;
					d_out[i].z -= 25;
				}
			}
		} 
		if (c % (w / (basamak * 5)) == 0){
			d_out[i].y -= 15;
			
		}
	}
}
__global__ void find2order(const  float* __restrict__  errorArray,
	uchar4 *d_out, int w, int h) {
	const int c = blockIdx.x*blockDim.x + threadIdx.x;
	const int r = blockIdx.y*blockDim.y + threadIdx.y;
	if ((c > 1) && (c < w - 1) && (r < h)) {
		int i = (c)+(h - r)*w; // 1D indexing
		float order = -errorArray[c - 1] + errorArray[c + 1];
		//order = logError(order) / (10);
		if (r < w / 2){
			if (order >= (float(r) / h)){
				d_out[i].x -= 20;
				d_out[i].y += 30;
			}
		}
	}
}

void error1D2D(float* accuracyTrain,
	float* accuracyTest,
	float* errorArray, uchar4 *out,
	int w, int h) {
	int block = 16;
	dim3 blockSize(block, block);
	dim3 gridSize((w + block - 1) / block, (h + block - 1) / block);
	error2TextureDevice << <gridSize, blockSize >> >(accuracyTrain, accuracyTest, errorArray, out, w, h);
	//printf("\nxx");
	//find2order << <gridSize, blockSize >> >( errorArray, out, w, h);
}




__global__ void errorSumDevice(const  float* __restrict__  paralelTensor,
	float * ErrorArray, int sizePiece, int step){
	 int Index = blockIdx.x*blockDim.x + threadIdx.x;
	// atomicAdd(&ErrorArray[step], (powf(errorLayer[Index], 2) / sizePiece));
	 atomicAdd(&ErrorArray[step], fabs(paralelTensor[Index]) / 500);
	//ErrorArray[step] += powf( errorLayer[Index] ,2)/ float(sizePiece);
 	// printf("\nooo  ooo :  %.10f", ErrorArray[step]);
}
void errorSummer(float* paralelTensor, float* errorArray,
	int sizePiece, int epoch, int widthScreen){
	if (epoch%widthScreen == 0){
		hipMemset(errorArray, 0, widthScreen* sizeof(float));
	}
	errorSumDevice << <sizePiece, 1 >> >(paralelTensor, errorArray, sizePiece, (epoch%widthScreen));
};



__global__ void errorPercentDe(const  float* __restrict__  outLayer,
	const  float* __restrict__ targetLayer,
	float * ErrorArray, int sampleTotal, int step){

	int Index = blockIdx.x*blockDim.x + threadIdx.x;
	if (targetLayer[Index]>0.5 && outLayer[Index] == 1.0){
		atomicAdd(&ErrorArray[step], 1.0 / sampleTotal);
	}
	else if (targetLayer[Index]< 0.5 && outLayer[Index] == 0.0){
		atomicAdd(&ErrorArray[step], 1.0 / sampleTotal);
	}
	
	//printf("\nooo  ooo :  out%.5f    target: %.5f ", outLayer[Index], targetLayer[Index]);
}

void errorPercent(float* OutLayer, float * targetLayer, float* errorArray,
	int sampleTotal,int sizePiece, int epoch,int widthScreen){
	if (epoch%widthScreen == 0){
		hipMemset(errorArray, 0, widthScreen* sizeof(float));
	}
	errorPercentDe << <sizePiece, 1 >> >(OutLayer, targetLayer, errorArray, sampleTotal, (epoch%widthScreen));
};



 __global__ void errorShow(const  float* __restrict__  Out,
	 const  float* __restrict__  target,int size){
	 int Index = blockIdx.x*blockDim.x + threadIdx.x;
	 printf("\nooo  ooo :  out%.5f    target: %.5f ", Out[Index], target[Index]);
 }
 void showCompare(float * outLay, float *target, int size){
	 errorShow << <size, 1 >> >(outLay, target, size);
 }
