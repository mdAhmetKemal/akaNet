#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "ConnectCuda.h"
#include <math.h>


__global__ void  ChangeFullyWeigthDe(const  float* __restrict__  inputWeLayerTensor,
	int inWeigthLaySamp,
	const  float* __restrict__  errorParalelTensor,
	float*  kernelParalelTensor, int inpFastSize, int outfastSize,
	float learning, float momentum, float* kernelWeigthTensor){
	int inputIndex, kernelIndex;
	int forSamp = blockIdx.x*blockDim.x + threadIdx.x;
	int numberOut = blockIdx.y*blockDim.y + threadIdx.y;
	float errorOutPiksel;
	
	if (forSamp < inWeigthLaySamp && numberOut < outfastSize)
	{
		errorOutPiksel = errorParalelTensor[inWeigthLaySamp*numberOut + forSamp];

		for (int fastInput = 0; fastInput < inpFastSize; fastInput++)
		{
			inputIndex = inWeigthLaySamp*(fastInput)+forSamp;
			kernelIndex = outfastSize*(fastInput)+numberOut;
			kernelParalelTensor[kernelIndex] += inputWeLayerTensor[inputIndex] * errorOutPiksel*learning;
		/*	printf("\n*** errorOutTarg: %.5f OutIndex %d  -inputX: %.5f  inInd %d  -kernelW: %.5f - KernelIndx: %d",
				errorOutPiksel, inWeigthLaySamp*numberOut + forSamp,
				inputWeLayerTensor[inputIndex], inputIndex,
				kernelWeigthTensor[kernelIndex], kernelIndex);
				*/
				
		}
	}
};
__global__ void  ChangeFullyWeBiasDe(
	int inWeigthLaySamp,
	const  float* __restrict__ errorParalelTensor,
	int outfastSize, float * biasParalelTensor,float learning,float momentum){

	int forSamp = blockIdx.x*blockDim.x + threadIdx.x;
	int numberOut = blockIdx.y*blockDim.y + threadIdx.y;
	float errorOutPiksel;

	if (forSamp < inWeigthLaySamp && numberOut < outfastSize)
	{
		//errorOutPiksel = errorParalelTensor[inWeigthLaySamp*numberOut + forSamp];
		biasParalelTensor[numberOut] += errorParalelTensor[inWeigthLaySamp*numberOut + forSamp] * learning;
		
	}
};
__global__ void  ChangeFullyWeUpdate(float* kernelWeigthTensor,
	const  float* __restrict__  kernelParalelTensor,
	int kernelFastSize){
	int kernelIndex = blockIdx.x*blockDim.x + threadIdx.x;
	if (kernelIndex < kernelFastSize){
	/*	printf("\n*** kernelW: %.4f -kernelDelW: %.6f - -KernelIndx:%d ",
			kernelWeigthTensor[kernelIndex],
			kernelParalelTensor[kernelIndex],
			kernelIndex); */
		kernelWeigthTensor[kernelIndex] = kernelWeigthTensor[kernelIndex] - (kernelParalelTensor[kernelIndex]);
	}
}

__global__ void  ChangeFullyWeMomentumDe(
	float*  kernelParalelTensor, int KernelFastSize,float momentum){
	int kernelIndex = blockIdx.x*blockDim.x + threadIdx.x;
	if (kernelIndex < KernelFastSize){
			kernelParalelTensor[kernelIndex] *= momentum;
			//printf("\nkernelParalelTensor[kernelIndex] : %.6f- kernelIndex %d ", kernelParalelTensor[kernelIndex], kernelIndex);
	}
}

void ChangeFullyWeigthCu(float* inputWeLayerTensor, int inWeigthLaySamp,
	int inWeigthLayFea, int inWeigthLayHe, int inWeigthLayWi, int inWeigthLayCh,
	float * ErrorOutParalelTensor, int errOutFe, int errOutHe, int errOutWi, int errOutCh,
	float* kernelWeigthTensor, float* kernelParalelTensor,
	float *biasTensor,
	float* biasParalelTensor,float learning,float momentum){
	
	int inpfastSize = inWeigthLayFea*inWeigthLayHe*inWeigthLayWi*inWeigthLayCh;
	int outFastSize = errOutFe*errOutHe*errOutWi*errOutCh;
	int kernelFastSize = outFastSize* inpfastSize;
	int blockNum = 256;
	dim3 BlockSize(blockNum);
	dim3 GridSize((kernelFastSize + BlockSize.x - 1) / BlockSize.x);
	ChangeFullyWeMomentumDe << <GridSize, BlockSize >> >(kernelParalelTensor, kernelFastSize, momentum);
	GridSize.x = (outFastSize + BlockSize.x - 1) / BlockSize.x;
	ChangeFullyWeMomentumDe << <GridSize, BlockSize >> >(biasParalelTensor, outFastSize, momentum);
	
	BlockSize.x = 2; BlockSize.y = 256;
	GridSize.x = (inWeigthLaySamp + BlockSize.x - 1) / BlockSize.x;
	GridSize.y = (outFastSize + BlockSize.y - 1) / BlockSize.y;
	ChangeFullyWeigthDe << < GridSize, BlockSize >> >(inputWeLayerTensor, inWeigthLaySamp,
		ErrorOutParalelTensor, kernelParalelTensor, inpfastSize, outFastSize, learning, momentum,
		kernelWeigthTensor);
	
	ChangeFullyWeBiasDe << < GridSize, BlockSize >> >(inWeigthLaySamp,ErrorOutParalelTensor,
		outFastSize,biasParalelTensor,learning,momentum);
	
	dim3 BlockSize2(256);
	dim3 GridSize2((kernelFastSize + BlockSize2.x - 1) / BlockSize2.x);
	ChangeFullyWeUpdate << < GridSize2, BlockSize2 >> >(kernelWeigthTensor, kernelParalelTensor,
		kernelFastSize);
	
	GridSize2.x = (outFastSize + BlockSize2.x - 1) / BlockSize2.x;
	ChangeFullyWeUpdate << < GridSize2, BlockSize2 >> >(biasTensor, biasParalelTensor,
		outFastSize);
};

