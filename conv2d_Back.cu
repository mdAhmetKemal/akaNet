#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "ConnectCuda.h"




__global__ void conv2d_BackCu(float*  inputWeParTensor, int inWeigthLaySamp,
	int inWeigthLayFea, int inWeigthLayHe, int inWeigthLayWi, int inWeigthLayCh,
	const  float* __restrict__ totalOutParTensor,
	int totalOutLayFe, int totalOutLayHe, int totalOutLayWi, int  totalOutLayCh,
	const  float* __restrict__ kernelWeigthTensor,
	int kernelFea, int kernelHeWeSc, int kernelDepth, int striX, int striY, int padX, int padY){
	int outHe = blockIdx.x*blockDim.x + threadIdx.x;
	int outWi = blockIdx.y*blockDim.y + threadIdx.y;
	int chanal = blockIdx.z*blockDim.z + threadIdx.z;
	if (outHe >= totalOutLayHe || outWi >= totalOutLayWi || chanal >= totalOutLayCh)
		return;

	int inputCentreIndex;
	int outPiks3dIndex = totalOutLayHe*(totalOutLayWi*(chanal)+outWi) + outHe;
	int halfKernelSize = (kernelHeWeSc - 1) / 2;
	int kernelIndex, inputIndex, outCentralIndex;
	int inHe = (outHe*striX) - padX - halfKernelSize;
	int inWi = (outWi*striY) - padY - halfKernelSize;
	for (int samNu = 0; samNu < inWeigthLaySamp; samNu++){
		for (int feaNum = 0; feaNum < inWeigthLayFea; feaNum++){

			for (int keFe = 0; keFe < kernelFea; keFe++){
				outCentralIndex = inWeigthLaySamp*(inWeigthLayFea*(kernelFea*(outPiks3dIndex)+keFe) + feaNum) + samNu;
				for (int keHe = 0; keHe < kernelHeWeSc; keHe++){
					for (int keWi = 0; keWi < kernelHeWeSc; keWi++){
						for (int keCha = 0; keCha < kernelDepth; keCha++){
							kernelIndex = totalOutLayFe*(kernelHeWeSc*(kernelHeWeSc*(keCha)+keWi) + keHe) + feaNum;
							inputCentreIndex = inWeigthLaySamp*(inWeigthLayFea*(inWeigthLayHe*(inWeigthLayWi*(chanal)+inWi + keHe) + inHe + keHe) + feaNum) + samNu;
							if (inHe < 0 || inHe >= totalOutLayHe || inWi < 0 || inWi >= totalOutLayWi){

							}
							else{
								inputWeParTensor[inputCentreIndex] += totalOutParTensor[outCentralIndex] * kernelWeigthTensor[kernelIndex];

							}
						}
					}
				}
			}

		}
	}
}

__global__ void _showouterror(
	const  float* __restrict__ erroroutparalel
	){
	int index = blockIdx.x*blockDim.x + threadIdx.x;


	printf("\n second %.10f   %d", erroroutparalel[index], index);
	//kernelWeigthTensor[kernelFea*kernelIndex + forFea] -= kernelParalelTensor[kernelFea*kernelIndex + forFea];

}
__global__ void _showinerror(
	const  float* __restrict__ erroroutparalel
	){
	int index = blockIdx.x*blockDim.x + threadIdx.x;


	printf("\n first %.10f   %d", erroroutparalel[index], index);
	//kernelWeigthTensor[kernelFea*kernelIndex + forFea] -= kernelParalelTensor[kernelFea*kernelIndex + forFea];

}

void conv2d_Back(float* inputWeLayerParTensor, int inWeigthLaySamp,
	int inWeigthLayFea, int inWeigthLayHe, int inWeigthLayWi, int inWeigthLayCh,
	float * totalOutLayerParTensor, int totalOutLayFe, int totalOutLayHe, int totalOutLayWi, int  totalOutLayCh,
	float* kernelWeigthTensor, int kernelFea, int kernelHeWeSc, int kernelDepth, int striX, int striY, int padX, int padY,
	float* biasTensor){
	int inpfastSize = inWeigthLaySamp*inWeigthLayFea*inWeigthLayHe*inWeigthLayWi*inWeigthLayCh;
	int outFastSize = inWeigthLaySamp*totalOutLayFe*totalOutLayHe*totalOutLayWi*totalOutLayCh;
	int blokX = 16;
	int blokY = 16;
	int blokZ = 2;
	dim3 BlockSize(blokX, blokY, blokZ);
	dim3 grido(inpfastSize);
	//_showinerror << <grido, 1 >> >(inputWeLayerParTensor);
	// burdansonras� iyi bir optimizasyon gerektirebilir
	dim3 GridSize((totalOutLayHe + BlockSize.x - 1) / BlockSize.x,
		(totalOutLayWi + BlockSize.y - 1) / BlockSize.y,
		(totalOutLayCh + BlockSize.z - 1) / BlockSize.z);
	hipMemset(inputWeLayerParTensor, 0, inpfastSize * sizeof(float));
	conv2d_BackCu << <GridSize, BlockSize >> >(inputWeLayerParTensor, inWeigthLaySamp,
		inWeigthLayFea, inWeigthLayHe, inWeigthLayWi, inWeigthLayCh,
		totalOutLayerParTensor,
		totalOutLayFe, totalOutLayHe, totalOutLayWi, totalOutLayCh,
		kernelWeigthTensor,
		kernelFea, kernelHeWeSc, kernelDepth, striX, striY, padX, padY);
	
	//_showinerror << <grido, 1 >> >(inputWeLayerParTensor);
	//_showinerror << <grido, 1 >> >(inputWeLayerParTensor);
};