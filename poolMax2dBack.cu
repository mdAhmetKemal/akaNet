#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "ConnectCuda.h"

__global__ void maxPoolBackCu(const  float* __restrict__ inputData, float* inputError, const  float* __restrict__  outputError, int sample, int feature,
	int chanal, int outHe, int outWi, int poolScale){
	int forHe = blockIdx.x*blockDim.x + threadIdx.x;
	int forWi = blockIdx.y*blockDim.y + threadIdx.y;
	float maxPiks;
	int maxX, maxY;
	if (forHe < outHe && forWi < outWi){
		for (int forSam = 0; forSam < sample; forSam++){
			for (int forFea = 0; forFea < feature; forFea++){
				for (int forCha = 0; forCha < chanal; forCha++){
					int outIndex = sample*(feature*(outHe*(outWi*(forCha)+forWi) + forHe) + forFea) + forSam;
					maxPiks = 0.0;
					maxX = 0; maxY = 0;
					for (int poolX = 0; poolX < poolScale; poolX++){
						for (int poolY = 0; poolY < poolScale; poolY++){
							//	printf("\n %.6f ", inputData[inIndex]);
							int inIndex = sample*(feature*(outHe*(outWi*(forCha)+(forWi*poolScale) + poolY) + (forHe*poolScale) + poolX) + forFea) + forSam;
							inputError[inIndex] = 0.0;
							if (inputData[inIndex] > maxPiks){ 
								maxX = poolX;
								maxY = poolY;
								maxPiks = inputData[inIndex];
							}
						}
					}
					inputError[sample*(feature*(outHe*(outWi*(forCha)+(forWi*poolScale) + maxY) + (forHe*poolScale) + maxX) + forFea) + forSam] =
						outputError[outIndex];
					//printf("\n******- %.12f   %d ", outputError[outIndex], outIndex);
				}
			}
		}
	}

}
__global__ void pool_showouterror(
	const  float* __restrict__ erroroutparalel
	){
	int index = blockIdx.x*blockDim.x + threadIdx.x;


	//printf("\n++^^ %.16f   %d", erroroutparalel[index], index);
	//kernelWeigthTensor[kernelFea*kernelIndex + forFea] -= kernelParalelTensor[kernelFea*kernelIndex + forFea];

}

void poolMax2dBack(Layer * inputLayer, Layer* outLayer, int poolScale){
	float * inputData = inputLayer->Tensor;
	float * inputError = inputLayer->paralelTensor;
	float * outputError = outLayer->paralelTensor;
	int inSam = inputLayer->hTsample;
	int inFea = inputLayer->hTfeatureNum;
	int inHe = inputLayer->hTheigth;
	int inWi = inputLayer->hTwidth;
	int inCha = inputLayer->hTchanal;
	int outSam = outLayer->hTsample;
	int outFea = outLayer->hTfeatureNum;
	int outHe = outLayer->hTheigth;
	int outWi = outLayer->hTwidth;
	int outCha = outLayer->hTchanal;
	//printf("\n poool  %d ", outLayer->sizePiksel);

	if (inSam == outSam && inFea == outFea  &&  inCha == outCha){
		if (inHe == outHe*poolScale  &&  inWi == outWi*poolScale){
			dim3 bloksize(16, 16);
			dim3 gridsize((outHe + bloksize.x - 1) / bloksize.x, (outWi + bloksize.y - 1) / bloksize.y);
			maxPoolBackCu << <gridsize, bloksize >> >(inputData,inputError, outputError, inSam, inFea, inCha, outHe, outWi, poolScale);
		}
		else printf("\n PoolLayer Scale Problem");
	}
	else printf("\n PoolLayer Scale Problem");
	//dim3 grido(inputLayer->sizePiksel);
	//pool_showouterror << <grido, 1 >> >(inputError);
}