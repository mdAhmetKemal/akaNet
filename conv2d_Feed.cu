#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "ConnectCuda.h"




__global__ void conv2d_FeedCu(const  float* __restrict__ inputWeLayerTensor, int inWeigthLaySamp,
	int inWeigthLayFea, int inWeigthLayHe, int inWeigthLayWi, int inWeigthLayCh,
	float * totalOutLayerTensor,
	int totalOutLayFe, int totalOutLayHe, int totalOutLayWi, int  totalOutLayCh,
	const  float* __restrict__ kernelWeigthTensor,
	int kernelFea, int kernelHeWeSc, int kernelDepth, int striX, int striY, int padX, int padY){
	int outHe = blockIdx.x*blockDim.x + threadIdx.x;
	int outWi = blockIdx.y*blockDim.y + threadIdx.y;
	int outCha = blockIdx.z*blockDim.z + threadIdx.z;
	if (outHe < totalOutLayHe && outWi < totalOutLayWi && outCha < totalOutLayCh){


		int outPiks3dIndex = totalOutLayHe*(totalOutLayWi*(outCha)+outWi) + outHe;
		int halfKernelSize = (kernelHeWeSc - 1) / 2;

		int outCentralIndex;
		int inHe, inWi;
		inHe = (outHe*striX) - padX - halfKernelSize;
		inWi = (outWi*striY) - padY - halfKernelSize;
		for (int samNu = 0; samNu < inWeigthLaySamp; samNu++){
			for (int feaNum = 0; feaNum < inWeigthLayFea; feaNum++){
				//hem out feature //hemde kernelFeature


				for (int keFe = 0; keFe < kernelFea; keFe++){
					outCentralIndex = inWeigthLaySamp*(inWeigthLayFea*(kernelFea*(totalOutLayHe*(totalOutLayWi*(outCha)
						+outWi) + outHe) + keFe) + feaNum) + samNu;
					for (int keHe = 0; keHe < kernelHeWeSc; keHe++){
						for (int keWi = 0; keWi < kernelHeWeSc; keWi++){
							for (int keCha = 0; keCha < kernelDepth; keCha++){
								if (inHe >= 0 && inHe < inWeigthLayHe && inWi >= 0 && inWi < inWeigthLayWi){
									totalOutLayerTensor[outCentralIndex] +=
										inputWeLayerTensor[inWeigthLaySamp*(inWeigthLayFea*(inWeigthLayHe*(inWeigthLayWi*(keCha)
										+inWi + keWi) + inHe + keHe) + feaNum) + samNu] *
										kernelWeigthTensor[kernelFea*(kernelHeWeSc*(kernelHeWeSc*(keCha)+keWi) + keHe) + keFe];
								}
								else{

								}
							}
						}
					}
				}
			}

		}
	}
}

__global__ void conv2d_FeedBias(
	int inpLaySamp, int totalOutFea, int kernelFea,
	float * outLayerTensor,
	int  outFastSize,
	const  float* __restrict__ biasData){

	int forSamp = blockIdx.x*blockDim.x + threadIdx.x;
	int forFea = blockIdx.y*blockDim.y + threadIdx.y;
	int kerFe = blockIdx.z*blockDim.z + threadIdx.z;
	if (forSamp < inpLaySamp && forFea < totalOutFea && kerFe<kernelFea){

		float bias = biasData[kerFe];
		for (int out = 0; out < outFastSize; out++){
			outLayerTensor[inpLaySamp*(totalOutFea*(kernelFea*(out)+kerFe) + forFea) + forSamp] += bias;
		}

	}
};

void conv2d_Feed(float* inputWeLayerTensor, int inWeigthLaySamp,
	int inWeigthLayFea, int inWeigthLayHe, int inWeigthLayWi, int inWeigthLayCh,
	float * totalOutLayerTensor, int totalOutLayFe, int totalOutLayHe, int totalOutLayWi, int  totalOutLayCh,
	float* kernelWeigthTensor, int kernelFea, int kernelHeWeSc, int kernelDepth, int striX, int striY, int padX, int padY,
	float* biasTensor){
	int inpfastSize = inWeigthLaySamp*inWeigthLayFea*inWeigthLayHe*inWeigthLayWi*inWeigthLayCh;
	int outFastSize = inWeigthLaySamp*totalOutLayFe*totalOutLayHe*totalOutLayWi*totalOutLayCh;

	if (inWeigthLayCh == kernelDepth){
		int blokX = 4;
		int blokY = 16;
		int blokZ = 2;
		dim3 BlockSize(blokX, blokY, blokZ);
		dim3 GridSize((totalOutLayHe + BlockSize.x - 1) / BlockSize.x,
			(totalOutLayWi + BlockSize.y - 1) / BlockSize.y,
			(totalOutLayCh + BlockSize.z - 1) / BlockSize.z);
		hipMemset(totalOutLayerTensor, 0, outFastSize * sizeof(float));
		conv2d_FeedCu << <GridSize, BlockSize >> >(inputWeLayerTensor, inWeigthLaySamp,
			inWeigthLayFea, inWeigthLayHe, inWeigthLayWi, inWeigthLayCh,
			totalOutLayerTensor,
			totalOutLayFe, totalOutLayHe, totalOutLayWi, totalOutLayCh,
			kernelWeigthTensor,
			kernelFea, kernelHeWeSc, kernelDepth, striX, striY, padX, padY);
		dim3 blok2(8, 8, 8);
		dim3 grid2((inWeigthLaySamp + BlockSize.x - 1) / BlockSize.x,
			(totalOutLayFe + BlockSize.y - 1) / BlockSize.y,
			(kernelDepth + BlockSize.z - 1) / BlockSize.z);
		outFastSize = totalOutLayHe*totalOutLayWi*totalOutLayCh;
	//	conv2d_FeedBias << <grid2, blok2 >> >(inWeigthLaySamp, totalOutLayFe, kernelFea, totalOutLayerTensor, outFastSize, biasTensor);
		//ActivationFunction(_RELU, totalOutLayerTensor, outFastSize);
	}
};