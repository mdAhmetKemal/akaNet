#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "ConnectCuda.h"

__global__ void avgPoolBackCu(const  float* __restrict__ inputData, float* inputError, const  float* __restrict__  outputError, int sample, int feature,
	int chanal, int outHe, int outWi, int poolScale){
	int forHe = blockIdx.x*blockDim.x + threadIdx.x;
	int forWi = blockIdx.y*blockDim.y + threadIdx.y;
	float maxPiks;
	int maxX, maxY;
	if (forHe < outHe &&forWi < outWi){
		for (int forSam = 0; forSam < sample; forSam++){
			for (int forFea = 0; forFea < feature; forFea++){
				for (int forCha = 0; forCha < chanal; forCha++){
					int outIndex = sample*(feature*(outHe*(outWi*(forCha)+forWi) + forHe) + forFea) + forSam;
					maxPiks = 0.0;
					for (int poolX = 0; poolX < poolScale; poolX++){
						for (int poolY = 0; poolY < poolScale; poolY++){
							//	printf("\n %.6f ", inputData[inIndex]);
							int inIndex = sample*(feature*(outHe*(outWi*(forCha)+forWi + poolY) + forHe + poolX) + forFea) + forSam;
							inputError[inIndex] = 0.0;
							if (inputData[inIndex] > maxPiks){
								maxX = poolX;
								maxY = poolY;
								//maxPiks = inputData[inIndex];
							}
						}
					}
					inputError[sample*(feature*(outHe*(outWi*(forCha)+forWi + maxY) + forHe + maxX) + forFea) + forSam] =
						outputError[outIndex];
					//printf("\n-%.6f ", outData[outIndex]);
				}
			}
		}
	}

}

void poolAvg2dBack(Layer * inputLayer, Layer* outLayer, int poolScale){
	float * inputData = inputLayer->Tensor;
	float * inputError = inputLayer->paralelTensor;
	float * outputError = outLayer->paralelTensor;
	int inSam = inputLayer->hTsample;
	int inFea = inputLayer->hTfeatureNum;
	int inHe = inputLayer->hTheigth;
	int inWi = inputLayer->hTwidth;
	int inCha = inputLayer->hTchanal;
	int outSam = outLayer->hTsample;
	int outFea = outLayer->hTfeatureNum;
	int outHe = outLayer->hTheigth;
	int outWi = outLayer->hTwidth;
	int outCha = outLayer->hTchanal;
	if (inSam == outSam && inFea == outFea  &&  inCha == outCha){
		if (inHe == outHe*poolScale  &&  inWi == outWi*poolScale){
			dim3 bloksize(16, 16);
			dim3 gridsize((outHe + bloksize.x - 1) / bloksize.x, (outWi + bloksize.y - 1) / bloksize.y);
			avgPoolBackCu << <gridsize, bloksize >> >(inputData, inputError, outputError, inSam, inFea, inCha, outHe, outWi, poolScale);
		}
		else printf("\n PoolLayer Scale Problem");
	}
	else printf("\n PoolLayer Scale Problem");
}