#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "ConnectCuda.h"




__global__ void flipKernelCu(const float* __restrict__ kernel3dData, float* flippedKernel3data, int TkerFea, int TkerHe, int TkerCha){
	int kerFea_N = blockIdx.x*blockDim.x + threadIdx.x;
	int kerHe_N = blockIdx.y*blockDim.y + threadIdx.y;
	int kerWi_N = blockIdx.z*blockDim.z + threadIdx.z;
	for (int kerCha_N = 0; kerCha_N < TkerCha; kerCha_N++){
		flippedKernel3data[TkerFea*(TkerHe*(TkerHe*(kerCha_N)+kerWi_N) + kerHe_N) + kerFea_N] =
			kernel3dData[TkerFea*(TkerHe*(TkerHe*(kerCha_N)+TkerHe - 1 - kerWi_N) + TkerHe - 1 - kerHe_N) + kerFea_N];
	}

}

__global__ void conv2dV2cu(const float * __restrict__ inputData, int totalSam, int TinFea, int TinHe, int TinWi, int TinCha,
	float * outData, int ToutFea, int ToutHe, int ToutWi, int ToutCha,
	const float* __restrict__ kernel3data, int TkerFea, int TkerHe, int TkerCha,
	int stride, int pad){

	int outHe_N = blockIdx.x*blockDim.x + threadIdx.x;
	int outWi_N = blockIdx.y*blockDim.y + threadIdx.y;
	int outCha_N = blockIdx.z*blockDim.z + threadIdx.z;
	if (outHe_N < ToutHe && outWi_N < ToutWi && outCha_N < ToutCha){

		int outMinIndex = ToutHe*(ToutWi*(outCha_N)+outWi_N) + outHe_N;
		int halfKerSize = (TkerHe - 1) / 2;
		int outGlobalIndex;
		int inHe_N = outHe_N*stride - pad - halfKerSize;
		int inWi_N = outWi_N*stride - pad - halfKerSize;
		int	inCha_N;
		for (int inSam_N = 0; inSam_N < totalSam; inSam_N++){
			for (int inFea_N = 0; inFea_N < TinFea; inFea_N++){
				for (int kerFea_N = 0; kerFea_N < TkerFea; kerFea_N++){
					outGlobalIndex = totalSam*(TinFea*(TkerFea*(ToutHe*(ToutWi*(outCha_N)+outWi_N) + outHe_N) + kerFea_N) + inFea_N) + inSam_N;
					for (int keHe_N = 0; keHe_N < TkerHe; keHe_N++){
						for (int keWi_N = 0; keWi_N < TkerHe; keWi_N++){
							for (int keCha_N = 0; keCha_N < TkerCha; keCha_N++){
								if (inHe_N  >= 0 && inHe_N  < TinHe && inWi_N >= 0 && inWi_N  < TinWi){
									outData[outGlobalIndex] =
										inputData[totalSam*(TinFea*(TinHe*(TinWi*(keCha_N/*inCha_N*/)+inWi_N + keWi_N) + inHe_N + keHe_N) + inFea_N) + inSam_N] *
										kernel3data[TkerFea*(TkerHe*(TkerHe*(keCha_N)+keWi_N) + keHe_N) + kerFea_N];
								}
							}
						}
					}
				}
			}
		}
	}
}



void conv2dV2Back(float* inputDifData, int totalSam,
	int inFea, int inHe, int inWi, int inCha,
	float * outputDifData, int outFea, int outHe, int outWi, int  outCha,
	float* kernelData, int kerFea, int kerHe, int kerCha, int striX, int striY, int padX, int padY,
	float* biasData) {
	if (inCha == kerCha){

		float * flipKernel;
		int kernelSize = kerFea*kerHe*kerHe*kerCha;
		hipMalloc(&flipKernel, kernelSize * sizeof(float));

		dim3 blockKernel(1,1,1);
		dim3 gridKernel(kerFea, kerHe, kerHe);
		flipKernelCu << <gridKernel, blockKernel >> >(kernelData, flipKernel,kerFea,kerHe,kerCha);

		int inputDataSize = totalSam*inCha*inHe*inWi*inCha;
		hipMemset(inputDifData, 0, inputDataSize * sizeof(float));
		int blokX = 16;
		int blokY = 16;
		int blokZ = 1;
		dim3 BlockSize(blokX, blokY, blokZ);
		dim3 GridSize(inHe / BlockSize.x + 1, inWi / BlockSize.y + 1, inCha);

		conv2dV2cu << <GridSize, BlockSize >> >(outputDifData, totalSam, outFea, outHe, outWi, outCha,
			inputDifData, inFea, inHe, inWi, inCha,
			flipKernel, kerFea, kerHe, kerCha, striX, padX);
		hipFree(flipKernel);
		//	conv2d_FeedBias << <grid2, blok2 >> >(inWeigthLaySamp, totalOutLayFe, kernelFea, totalOutLayerTensor, outFastSize, biasTensor);
		//ActivationFunction(_RELU, totalOutLayerTensor, outFastSize);
	}
};
void conv2dV2Feed(float* inputData, int totalSam,
	int inFea, int inHe, int inWi, int inCha,
	float * outputData, int outFea, int outHe, int outWi, int  outCha,
	float* kernelData, int kerFea, int kerHe, int kerCha, int striX, int striY, int padX, int padY,
	float* biasData) {
	int outputDataSize = totalSam*outFea*outHe*outWi*outCha;
	if (inCha == kerCha){
		hipMemset(outputData, 0, outputDataSize * sizeof(float));
		int blokX = 16;
		int blokY = 16;
		int blokZ = 1;
		dim3 BlockSize(blokX, blokY, blokZ);
		dim3 GridSize(outHe / BlockSize.x + 1, outWi / BlockSize.y + 1, outCha);

		conv2dV2cu << <GridSize, BlockSize >> >(inputData, totalSam, inFea, inHe, inWi, inCha,
			outputData, outFea, outHe, outWi, outCha,
			kernelData, kerFea, kerHe, kerCha, striX, padX);

		//	conv2d_FeedBias << <grid2, blok2 >> >(inWeigthLaySamp, totalOutLayFe, kernelFea, totalOutLayerTensor, outFastSize, biasTensor);
		//ActivationFunction(_RELU, totalOutLayerTensor, outFastSize);
	}
};