#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "ConnectCuda.h"
#include <hip/hip_runtime.h>
/*
__global__ void errorSoftmaxtargetDiv(const  float* __restrict__ outLayer,
	const  float* __restrict__ targetLayer, float* __restrict__ sampleErrorSumExpo,
	int sampleNum,int totalSize){
	int samplNumIn = blockIdx.x ;
	int Index =  threadIdx.x;
	//printf("\n Index %d", Index);
	if (samplNumIn<sampleNum){
		if (Index < totalSize){
			if (targetLayer[samplNumIn*totalSize + Index] == 1.){
				//printf("\n out%.6f  calc:%.5f      res:%.5f     inde:%d", expf(outLayer[samplNumIn*totalSize + Index]), sampleErrorSumExpo[samplNumIn],
					//targetLayer[samplNumIn*totalSize + Index], samplNumIn*totalSize + Index);
				sampleErrorSumExpo[samplNumIn] = expf(outLayer[samplNumIn*totalSize + Index]) / sampleErrorSumExpo[samplNumIn];
				//printf("\n out%.6f  calc:%.5f      res:%.5f     inde:%d", expf(outLayer[samplNumIn*totalSize + Index]), sampleErrorSumExpo[samplNumIn],
					//targetLayer[samplNumIn*totalSize + Index], samplNumIn*totalSize + Index);
			}
			
		}
	}
}

__global__ void errorSoftmaxSumExpo(const  float* __restrict__  outLayer,
	float * sampleErrorSumExpo, int sampleNum, int totalPikselSize){
	int samplNumIn = blockIdx.x*blockDim.x + threadIdx.x;
	int Index = blockIdx.y*blockDim.y + threadIdx.y;
	if (samplNumIn < sampleNum){
		atomicAdd(&sampleErrorSumExpo[samplNumIn], expf(outLayer[samplNumIn*totalPikselSize + Index]));
		//printf("\nooo:  %.4f  exp%.4f", sampleErrorSumExpo[samplNumIn], expf(outLayer[samplNumIn*totalPikselSize + Index]) );
	}
	//ErrorArray[step] += powf( errorLayer[Index] ,2)/ float(sizePiece);
	
} */

__global__ void errorSoftmaxGenelalize(float* __restrict__ sampleErrorSumExpo,
	float * outParalelTensor,
	int sampleNum, int totalSize){
	int samplNumIn = blockIdx.x*blockDim.x + threadIdx.x;
	int Index = blockIdx.y*blockDim.y + threadIdx.y;
	if (samplNumIn < sampleNum){
		if (Index < totalSize){
			outParalelTensor[samplNumIn*totalSize + Index] = sampleErrorSumExpo[samplNumIn];
		//	printf("\nooo:  %.4f  exp%.4f  in:%d", outParalelTensor[samplNumIn*totalSize + Index], sampleErrorSumExpo[samplNumIn],Index);
		}
	}
}


__global__ void errorSoftmaxModify(const  float* __restrict__ outLayer,
	const  float* __restrict__ targetLayer, float* __restrict__ sampleErrorSumExpo,
	float* __restrict__ outParalelTensor,
	int sampleNum, int totalPikselSize){
	int samplNumIn = blockIdx.x*blockDim.x + threadIdx.x;
	float totalExpo = 0;
	float totalError = 0;
	float tempSoft = 0;
	float temptarget = 0;
	float tempOut = 0;
	float tempError = 0;
	if (samplNumIn < sampleNum){
		for (int p = 0; p < totalPikselSize; p++){
			totalExpo += expf(outLayer[samplNumIn*totalPikselSize + p]);
		}
	
		for (int p = 0; p < totalPikselSize; p++){
			tempOut = outLayer[samplNumIn*totalPikselSize + p];//calculateed
			tempSoft = expf(tempOut) / totalExpo;
			temptarget = targetLayer[samplNumIn*totalPikselSize + p];//true Labels
			//tempError = -1*((temptarget*logf(tempSoft) + (1 - temptarget)*logf(1.00000 - tempSoft)));
			//if (tempOut < 0) tempOut = 0;
			//a�a��s� harika �al���yor
			//tempError = -1 * ((temptarget*(temptarget - tempSoft)) + ((1 - temptarget)*(temptarget - tempSoft)));
			tempError = -1*(temptarget - tempSoft);
			//totalError += fabs(tempError);
			outParalelTensor[samplNumIn*totalPikselSize + p] = tempError;
			//printf("\n--  output:%.4f  %.2f :targ    error%.4f  ",tempOut,
				// temptarget, outParalelTensor[samplNumIn*totalPikselSize + p]);
		}
		/*for (int p = 0; p < totalPikselSize; p++){
			totalExpo += expf(outLayer[samplNumIn*totalPikselSize + p]);
			trueValExp += expf((outLayer[samplNumIn*totalPikselSize + p])*targetLayer[samplNumIn*totalPikselSize + p])+
				((1-outLayer[samplNumIn*totalPikselSize + p])*(1-targetLayer[samplNumIn*totalPikselSize + p]) );
			printf("\n----expOut %.6f   res:%.6f in:%d", expf(outLayer[samplNumIn*totalPikselSize + p]),
				targetLayer[samplNumIn*totalPikselSize + p],p);
		}
*/
	//	sampleErrorSumExpo[samplNumIn] = totalError/totalPikselSize;
	//	printf("\n error:%.4f  ", totalError / totalPikselSize);
	
	}
}

void errorSoftmaxCrEn(float * outLayer, float* targetLayer, float * outParalelTensor, int sampleNum, int outFea,
	int outHe, int outWi, int outCha){
	float * sampleErrorSumExpo;
	hipMalloc(&sampleErrorSumExpo, sampleNum * sizeof(float));
	hipMemset(sampleErrorSumExpo, 0, sampleNum * sizeof(float));
	int totalPikselSize =  outFea*outHe*outWi*outCha;
	int block = 8;
	dim3 BlockSize(block);
	dim3 GridSize((sampleNum + block - 1) / BlockSize.x);
	errorSoftmaxModify << <GridSize, BlockSize >> >(outLayer, targetLayer, sampleErrorSumExpo, outParalelTensor, sampleNum, totalPikselSize);
	//printf("\n SOftmax totalpiksel %d", totalPikselSize);
	//errorSoftmaxtargetDiv <<<sampleNum, totalPikselSize >> >(outLayer, targetLayer, sampleErrorSumExpo, sampleNum, totalPikselSize);
	BlockSize.x = 8; BlockSize.y = 32;
	GridSize.x = (sampleNum + BlockSize.x - 1) / BlockSize.x;
	GridSize.y = (totalPikselSize + BlockSize.y - 1) / BlockSize.y;
//	errorSoftmaxGenelalize << <GridSize, BlockSize >> >(sampleErrorSumExpo, outParalelTensor, sampleNum, totalPikselSize);
	hipFree(sampleErrorSumExpo);
}


///////****************************** 
/*

softmax d�zg�n �al���yor mu ayr�ca negatif log unutma 

Netdata fonnksiyonu son out datas� nominal ise t�pk� di�erlerinde oldu�u gibi bir s�n�fland�rma yapm�yor
ikili ��l� s�n�fland�rma yerine sadece 1 2 diyor sorun da bu 



*/
