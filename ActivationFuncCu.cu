#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "ConnectActFunc.h"
#include <math.h>
#include <hip/hip_runtime.h>

__global__ void lineerAct(float * layer, int layerSize){

}
__global__ void ReluAct(float * layer, int layerSize){
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index < layerSize){
		if (layer[index] < 0){
			layer[index] = 0.0;
		}
	//	else{
			// layer[index] = layer[index];
	//	}
	}
}
__global__ void LReluAct(float * layer, int layerSize, float parameter){
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index < layerSize){
		if (layer[index] < 0){
			layer[index] = layer[index] * parameter;
		}
	//	else{
		//	layer[index] = layer[index];
	//	}
	}
}
__global__ void EluAct(float * layer, int layerSize,float parameter){
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index < layerSize){
		if (layer[index] < 0){
			layer[index] = parameter*(expf(layer[index]) - 1);
		}
		//	else{
		// layer[index] = layer[index];
		//	}
	}
}
__global__ void SigmAct(float * layer, int layerSize){
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index < layerSize){
			layer[index] = 1/(expf(-layer[index]) + 1);	
	}
}
__global__ void TanhAct(float * layer, int layerSize){
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index < layerSize){
		layer[index] = (2 / (expf(-2 * layer[index]) + 1)) - 1;
	}
}
__global__ void StepAct(float * layer, int layerSize){
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index < layerSize){
		if (layer[index] < 0){
			layer[index] = 0.0;
		}
			else{
		 layer[index] = 1.0;
		}
	}
}
__global__ void ArctanAct(float * layer, int layerSize){
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index < layerSize){
		layer[index] = atanf( layer[index]);
	}
}
__global__ void SoftplusAct(float * layer, int layerSize){
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index < layerSize){
		layer[index] = log1pf(expf(layer[index])) ;
	}
}
extern void ActivationFunction(_ConnectActType layerAct, float * layer, int layerSize,float parameter){
	int block = 512;
	dim3 BlockSize(block);
	dim3 GridSize(layerSize + BlockSize.x - 1 / BlockSize.x);
	if (layerAct == _NOACTV || layerAct == _LINEER){
		return;
	}
	if (layerAct == _RELU){
		ReluAct << <GridSize, BlockSize >> >(layer, layerSize);
	}
	if (layerAct == _ELU){
		EluAct << <GridSize, BlockSize >> >(layer, layerSize, parameter);
	}
	if (layerAct == _LRELU){
		LReluAct << <GridSize, BlockSize >> >(layer, layerSize,parameter);
	}
	if (layerAct == _SIGM){
		SigmAct << <GridSize, BlockSize >> >(layer, layerSize);
	}
	if (layerAct == _TANH){
		TanhAct << <GridSize, BlockSize >> >(layer, layerSize);
	}
	if (layerAct == _STEP){
		StepAct << <GridSize, BlockSize >> >(layer, layerSize);
	}
	if (layerAct == _ARCTAN){
		ArctanAct << <GridSize, BlockSize >> >(layer, layerSize);
	}
	if (layerAct == _SOFTPLUS){
		SoftplusAct << <GridSize, BlockSize >> >(layer, layerSize);
	}
}