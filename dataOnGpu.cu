#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "ConnectCuda.h"




__global__ void shuffleGpuCu(const float* __restrict__ sourceData, float * shuffledData,int sampleNum,
	int sampleSize,  int*  shuffleArray){
	int step = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (step < sampleNum){
		int tempStep = shuffleArray[step];
		for (int a = 0; a < sampleSize; a++){
			shuffledData[sampleNum*a + step] = sourceData[sampleNum*a + tempStep];
		}
		
	}
};

void shuffleGpu(float* sourceData,float * shuffledData, int totalSample, int heigth,
	int width,int chanal, int * shuffleArray){

	int * arrayGpu;
	hipError_t target = hipMalloc(&arrayGpu, totalSample*sizeof(int));
	hipMemcpy(arrayGpu, shuffleArray, totalSample*sizeof(int), hipMemcpyHostToDevice);

	dim3 Bloksize(512);
	dim3 Gridsize(totalSample / Bloksize.x + 1);

	shuffleGpuCu << <Gridsize, Bloksize >> >(sourceData, shuffledData, totalSample,
		heigth*width*chanal, arrayGpu);
	hipFree(arrayGpu);

}


__global__ void datashowCu(float * data, int size,int sample){

	for (int b = 0; b < sample; b++){
		for (int a = 0; a < size; a++){

			if (data[sample*(a) + b] > 0){
				printf(" \x6A");
			}
			else{
				printf("  ");
			}

			if ((a + 1) % 28 == 0){
				printf("\n");
			}
		}
	}
}
void showonGpuData(float * sourceData, int size, int totalsample){
	printf("\n  %d  %d   \n", size, totalsample);
	if (size == 784){
		datashowCu << <1, 1 >> >(sourceData, size, totalsample);
	}
}


__global__ void partialSet(const float* __restrict__ sourceData, float * targetData, int sample, int size, int piece, int iSamTotal){

	for (int forSamp = 0; forSamp < sample; forSamp++){
		for (int forSiz = 0; forSiz < size; forSiz++){
			targetData[sample*(forSiz)+forSamp] = sourceData[iSamTotal*(forSiz)+forSamp + piece];
		}
	}
}


void loadPieceOnGpu(float * sourceData, float* loadedData, int heigth, int totalsample, int ProcesStepSample, int inputSampleTotal){
	partialSet << <1, 1 >> >(sourceData, loadedData, totalsample, heigth, ProcesStepSample*totalsample, inputSampleTotal);
}
