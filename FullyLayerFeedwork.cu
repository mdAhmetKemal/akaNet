#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "ConnectCuda.h"

__global__ void  FullyLayerFeedworkDeNoShaAlterna(
	const  float* __restrict__ inputWeLayerTensor,
	int inWeigthLaySamp,
	float * totalOutLayerTensor, const  float* __restrict__ kernelWeigthTensor,
	int inputfastSize, int kernelFastSize, int outFastSize,
	const  float* __restrict__ biasTensor){
	int forSamp = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int numberOut = blockIdx.y*blockDim.y + threadIdx.y;

	if (forSamp < inWeigthLaySamp && numberOut < outFastSize){
		float outPiksel = 0.;
		int kernelIndex, inputIndex;
#pragma unroll 
			for (int fastInput = 0; fastInput < inputfastSize; fastInput++)
			{
				inputIndex = inWeigthLaySamp*(fastInput)+forSamp;
				kernelIndex = outFastSize*(fastInput)+numberOut;
				outPiksel += inputWeLayerTensor[inputIndex] * kernelWeigthTensor[kernelIndex];
	/*		printf("\n-->Flfeed  in inPiVa : %.5f - inInd %d  kernelWe: %.5f - keInd %d   outPiVa:%.5f - outInd %d ",
					inputWeLayerTensor[inputIndex], inputIndex,
					kernelWeigthTensor[kernelIndex], kernelIndex,
					outPiksel,(inWeigthLaySamp*numberOut + forSamp)); */
			}
			totalOutLayerTensor[inWeigthLaySamp*numberOut + forSamp] = outPiksel;// +biasTensor[numberOut];
		/*	printf("\n>Flfeed outPiNoBi : %.5f - outWitBias %.5f  outIndx: - %d  bias: %.5f -biasIndx %d ", outPiksel,
				totalOutLayerTensor[inWeigthLaySamp*numberOut + forSamp],
				(inWeigthLaySamp*numberOut + forSamp),
				 biasTensor[numberOut], numberOut); */
	}
};
__global__ void  FullyBiasAdd(
	int inpLaySamp,
	float * outLayerTensor,
	int  outFastSize,
	const  float* __restrict__ biasData){

	int forSamp = blockIdx.x*blockDim.x + threadIdx.x;
	int numberOut = blockIdx.y*blockDim.y + threadIdx.y;
	if (forSamp < inpLaySamp && numberOut < outFastSize){
		outLayerTensor[inpLaySamp*numberOut + forSamp] += biasData[numberOut];
	}
};
void FullyLayerFeedworkCu(
	float* inputWeLayerTensor, int inWeigthLaySamp, int inWeigthLayFea,
	int inWeigthLayHe, int inWeigthLayWi, int inWeigthLayCh, 
	float * totalOutLayerTensor,int totalOutLayFe, int totalOutLayHe, int totalOutLayWi,
	int totalOutLayCh,
	float* kernelWeigthTensor, float* biasTensor){
	int inpfastSize = inWeigthLayFea*inWeigthLayHe*inWeigthLayWi*inWeigthLayCh;
	int outFastSize = totalOutLayFe*totalOutLayHe*totalOutLayWi*totalOutLayCh;
	int kernelFastSize = outFastSize* inpfastSize;

	dim3 BlockSize(2, 512);
	dim3 GridSize((inWeigthLaySamp + 2 - 1) / BlockSize.x, (outFastSize + 512 - 1) / BlockSize.y);
	FullyLayerFeedworkDeNoShaAlterna << < GridSize, BlockSize >> >(inputWeLayerTensor, inWeigthLaySamp,
		totalOutLayerTensor,
		kernelWeigthTensor, inpfastSize, kernelFastSize, outFastSize, biasTensor);
	FullyBiasAdd << < GridSize, BlockSize >> >
		(inWeigthLaySamp, totalOutLayerTensor, outFastSize, biasTensor);
//	ActivationFunction(_RELU, totalOutLayerTensor, outFastSize);
};

/*buna benzer fully connect witt activitation i�in  4-4-3 boyurlu input-regulator-output  layerlar�ndan
olu�an 3 ba�lant�l� konnect alan ve input-inputweigth-regulator-regulatorActivite-output datalar� olan bir 
connect s�n�f� laz�m bunlar� i�lem yaparken �zellikle input -regulator k�sm� i�in shared meymory kullanma 
�ans�n� kullanmak gerekli olacak gibi durutyor */
/*
__global__ void  FullyLayerFeedworkDeNoShare(
const  float* __restrict__ inputWeLayerTensor,
int inWeigthLaySamp, int inWeigthLayFea,
int inWeigthLayHe, int inWeigthLayWi, int inWeigthLayCh,
float * totalOutLayerTensor, int totalOutLayHe, int totalOutLayWi,
int totalOutLayCh, const  float* __restrict__ kernelWeigthTensor,
int forSamp,int outHe,int outWi,int outCh){
int inHe = blockIdx.x*blockDim.x + threadIdx.x;
int inWi = blockIdx.y*blockDim.y + threadIdx.y;
int inFe = blockIdx.z*blockDim.z + threadIdx.z;

if (inHe < inWeigthLayHe && inWi < inWeigthLayWi && inFe < inWeigthLayFea){
int inputIndex, kernelIndex;
float outPiksel = 0;
for (int inCh = 0; inCh < inWeigthLayCh; inCh++)
{
//inputIndex = inWeigthLaySamp*(inWeigthLayFea*(inWeigthLayHe*(inWeigthLayWi*(inCh)+inWi) + inHe) + inFe) + forSamp;
//kernelIndex = totalOutLayHe*(totalOutLayWi*(totalOutLayCh*(inWeigthLayFea*(inWeigthLayHe*(inWi)
//	+inHe) + inFe)+outCh) + outWi) + outHe;
//outPiksel += inputWeLayerTensor[inputIndex] * kernelWeigthTensor[kernelIndex];
outPiksel += inputWeLayerTensor[inWeigthLaySamp*(inWeigthLayFea*(inWeigthLayHe*(inWeigthLayWi*(inCh)+inWi) + inHe) + inFe) + forSamp] *
kernelWeigthTensor[totalOutLayHe*(totalOutLayWi*(totalOutLayCh*(inWeigthLayFea*(inWeigthLayHe*(inWi)+inHe) + inFe)+outCh) + outWi) + outHe];
}
totalOutLayerTensor[inWeigthLaySamp*(totalOutLayHe*(totalOutLayWi*(outCh)+outWi) + outHe) + forSamp] = outPiksel;
}
};

__global__ void  FullyLayerFeedworkDeShared(
const  float* __restrict__ inputWeLayerTensor,
int inWeigthLaySamp, int inWeigthLayFea,
int inWeigthLayHe, int inWeigthLayWi, int inWeigthLayCh,
float * totalOutLayerTensor, int totalOutLayHe, int totalOutLayWi,
int totalOutLayCh, const  float* __restrict__ kernelWeigthTensor,
int inputfastSize, int kernelFastSize, int outFastSize){
int forSamp = blockIdx.x*blockDim.x + threadIdx.x;
int numberOut = blockIdx.y*blockDim.y + threadIdx.y;
//int numberOut = inWeigthLaySamp*(totalOutLayHe*(totalOutLayWi*(outCh)+outWi) + outHe) + forSamp;
__shared__  float inputFastData[1000];

if (forSamp < inWeigthLaySamp && numberOut < outFastSize){

for (int inputIndex = 0; inputIndex < inputfastSize; inputIndex++){
inputFastData[inputIndex] = inputWeLayerTensor[inWeigthLaySamp*(inputIndex)+forSamp];
__syncthreads();
}

float outPiksel = 0;
int kernelIndex, inputIndex;
for (int fastInput = 0; fastInput < inputfastSize; fastInput++)
{
#pragma unroll inWeigthLaySamp
kernelIndex = outFastSize*(fastInput)+numberOut;
//printf("\nkernel %d", kernelIndex);
outPiksel += inputFastData[fastInput] * kernelWeigthTensor[kernelIndex];
}
//printf("out index %d", numberOut);
totalOutLayerTensor[numberOut] = outPiksel;
}
};
*/