#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"
#include <hip/hip_runtime.h>
#include "HyperTensor.h"

__global__ void  randTensorD(float * data,int size,float fac,unsigned int seed){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < size){
		hiprandState states;
		hiprand_init(i, 1, 0, &states);
		data[i] = (hiprand_uniform(&states) * fac) - (fac / float(2));
		//printf("val:%.5f  ",  data[i]);
	}
};
__global__ void TensorFactor(float * data,const int size,const float fac){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < size){
		data[i] = (data[i]*fac)-(fac/float(2));
		
	}
};


void  randTensor(float* data, int size,float fac){

	//printf("\nCu:randtensor fac:%.4f", fac);
	int block = 32;
	dim3 BlockSize(block);
	dim3 GridSize((size + block - 1) / BlockSize.x);
	randTensorD << < GridSize, BlockSize >> >(data,size,fac, 13142737);

};