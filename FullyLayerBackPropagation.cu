#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "ConnectCuda.h"


__global__ void  FullyLayerBackPropagationDeShared(float*  inputParalelTensor,
	int inWeigthLaySamp, int inWeigthLayFea,
	int inWeigthLayHe, int inWeigthLayWi, int inWeigthLaySize,
	const  float* __restrict__  errorParalelTensor, int totalOutLayHe, int totalOutLayWi, int totalOutLaySize,
	const  float* __restrict__ kernelWeigthTensor){
	int outIndex, kernelIndex, inputIndex;
	float inPiksel;
	const int fastOutSize = totalOutLayHe*totalOutLayWi;
	__shared__  float outFastData[10100];
	int idxHe = blockIdx.x*blockDim.x + threadIdx.x;
	int idxWi = blockIdx.y*blockDim.y + threadIdx.y;
	int idxFea = blockIdx.z*blockDim.z + threadIdx.z;
	int totalPiks = inWeigthLayFea*(inWeigthLayHe*(inWeigthLayWi*idxWi) + idxHe) + idxFea;
	if (totalPiks < inWeigthLaySize)
	{
		for (int inSam = 1; inSam < inWeigthLaySamp; inSam++)
		{
			inPiksel = 0;
			for (int fast = 0; fast < fastOutSize; fast++)
				for (int errHe = 1; errHe < totalOutLayHe; errHe++)
				{
					for (int errWi = 1; errHe < totalOutLayWi; errWi++)
					{
						outIndex = inWeigthLaySamp*(totalOutLayHe*(errWi)+errHe) + inSam;
						outFastData[fast] = errorParalelTensor[outIndex];
						__syncthreads();
					}
				}
			
			inputIndex = inWeigthLaySamp*(inWeigthLayFea*(inWeigthLayHe*(idxWi)+idxHe) + idxFea) + inSam;
			for (int errHe = 1; errHe < totalOutLayHe; errHe++)
			{
				for (int errWi = 1; errWi < totalOutLayWi; errWi++)
				{
					outIndex = totalOutLayHe*(errWi)+errHe;
					kernelIndex = inWeigthLayFea*(inWeigthLayHe*(inWeigthLayWi*(totalOutLayHe*(errWi)+errHe) + idxWi) + idxHe) + idxFea;
					inPiksel += outFastData[outIndex] * kernelWeigthTensor[kernelIndex];
					__syncthreads();
				}
			}
		
			inputParalelTensor[inputIndex] = inPiksel;
		}
	}
};

__global__ void  FullyLayerBackPropagationDeNoshared(float*  inputParalelTensor,
	int inWeigthLaySamp,
	const  float* __restrict__  errorParalelTensor, 
	const  float* __restrict__ kernelWeigthTensor, int inpFastSize,int KernelFastSize,int outfastSize){
	int errOutIndex, kernelIndex;
	float errInPiksel;
	int forSamp = blockIdx.x*blockDim.x + threadIdx.x;
	int numberInp = blockIdx.y*blockDim.y + threadIdx.y;

	if (forSamp < inWeigthLaySamp &&numberInp < inpFastSize)
	{
		errInPiksel = 0.;
#pragma unroll 
		for (int fastOut = 0; fastOut < outfastSize; fastOut++)
		{
			//kernelIndex = outFastSize*(fastInput)+numberOut;
			errOutIndex = inWeigthLaySamp*(fastOut)+forSamp;
			kernelIndex = inpFastSize*(fastOut)+numberInp;
			errInPiksel += errorParalelTensor[errOutIndex] * kernelWeigthTensor[kernelIndex];
		/*	printf("\n-->FlBackProfast out InPiksel : %.6f %d  kernelWe: %.6f toplam errorIn:%.6f", errorParalelTensor[errOutIndex],
				errOutIndex,
				kernelWeigthTensor[kernelIndex],
				kernelIndex, errInPiksel); */
		}
		inputParalelTensor[inWeigthLaySamp*numberInp + forSamp] = errInPiksel;
	//	printf("\n>FlBaP  : %.6f  inputIndex %d ", inputParalelTensor[inWeigthLaySamp*numberInp + forSamp],
		//	inWeigthLaySamp*numberInp + forSamp); 
	} 
};

void FullyLayerBackPropagationCu(float* inputErrorTensor, int inErrorLaySamp, int inErrorLayFea,
	int inErrorLayHe, int inErrorLayWi, int inErrorLayCh,
	float * errorOutLayerTensor, int totalOutLayFe, int totalOutLayHe, int totalOutLayWi, int totalOutLayCh,
	float* kernelWeigthTensor){
	int inpfastSize = inErrorLayFea*inErrorLayHe*inErrorLayWi*inErrorLayCh;
	int outFastSize = totalOutLayFe*totalOutLayHe*totalOutLayWi*totalOutLayCh;
	int kernelFastSize = outFastSize* inpfastSize;
	dim3 BlockSize(2, 512);
	dim3 GridSize((inErrorLaySamp + 2 - 1) / BlockSize.x, (inpfastSize + 512 - 1) / BlockSize.y);

	FullyLayerBackPropagationDeNoshared << < GridSize, BlockSize >> >(inputErrorTensor, inErrorLaySamp,
		errorOutLayerTensor, kernelWeigthTensor, inpfastSize, kernelFastSize, outFastSize);
	//DerivationFunction(_RELU, inputErrorTensor, inpfastSize);
};
