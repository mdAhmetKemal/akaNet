#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "ConnectCuda.h"

__global__ void layerShowerCuda(uchar4 * other_out,int pikSize,int W,int H,const float * __restrict__ layer,
	int layerSample,int layerFeature,int layerChanal,int layerHeigth,int layerWeigth,
	int maxSample,int maxFeature){
	const int col = blockIdx.x*blockDim.x + threadIdx.x;
	const int row = blockIdx.y*blockDim.y + threadIdx.y;
	const int cha = blockIdx.z*blockDim.z + threadIdx.z;
	int thisSample, thisFeature,thisHeigth,thisWeigth;
	
	if (col < W && row < H){
		
		int outIndex = H*(row+6)+col+2;
		if (row < (H/2)&& col<(W/1)){
		
			thisSample = (col / (pikSize*layerWeigth));
			thisWeigth  = (col % (pikSize*layerWeigth)) / pikSize;
			thisFeature = (row / (pikSize*layerHeigth));
			thisHeigth = (row % (pikSize*layerHeigth)) / pikSize;
			if (thisSample < maxSample && thisFeature < maxFeature){
				//int inIndex = (layerWeigth *(layerHeigth *(layerSample*(0) + thisSample) + thisHeigth) + thisWeigth);
				int inIndex = layerSample*(layerFeature*(layerHeigth*(layerWeigth*(cha) +thisWeigth) + thisHeigth) + thisFeature) + thisSample;
				other_out[outIndex].x = unsigned(layer[inIndex] * 255);
				other_out[outIndex].y = unsigned(layer[inIndex] * 255);
				other_out[outIndex].z = unsigned(layer[inIndex] * 255);
				other_out[outIndex].w = 255;
				//*******sorunlar� burdaki g�stergecler ile bulabilirsin 
				if (false){
					printf("\n lS:%d tS:%d mS:%d lF:%d tF:%d mF:%d lH:%d tH:%d lW:%d tW:%d col:%d row:%d H:%d W:%d piSi:%d ",
						layerSample, thisSample, maxSample, layerFeature, thisFeature, maxFeature, layerHeigth, thisHeigth, layerWeigth,
						thisWeigth, col, row, H, W, pikSize);
				}
					
			} else{
			other_out[outIndex].x = unsigned(240);
			other_out[outIndex].y = unsigned(240);
			other_out[outIndex].z = other_out[outIndex].x;
			other_out[outIndex].w = 255;
			}
			/*  eskisi 
			thisSample = (row / (pikSize*layerHeigth));
			thisWeigth = row % (pikSize*layerHeigth);
			thisFeature = (col / (pikSize*layerWeigth));
			thisHeigth = col % (pikSize*layerWeigth);
			if (thisSample < maxSample && thisFeature < maxFeature){
				int inIndex = layerSample*(layerFeature*(layerHeigth*(layerWeigth*(cha)+thisWeigth) + thisHeigth) + thisFeature) + thisSample;
				other_out[outIndex].x = unsigned(layer[inIndex] * 256);
				other_out[outIndex].y = other_out[outIndex].x;
				other_out[outIndex].z = other_out[outIndex].x;
				other_out[outIndex].w = 255;
				//*******sorunlar� burdaki g�stergecler ile bulabilirsin 
				 printf("\n lS:%d tS:%d mS:%d lF:%d tF:%d mF:%d lH:%d tH:%d lW:%d tW:%d col:%d row:%d H:%d W:%d piSi:%d ",
					layerSample, thisSample, maxSample, layerFeature, thisFeature, maxFeature, layerHeigth, thisHeigth, layerWeigth,
					thisWeigth, col, row, H, W, pikSize); 
			}
		*/
			
		}
		else
		{
			
			other_out[outIndex].x = unsigned(240);
			other_out[outIndex].y = unsigned(240);
			other_out[outIndex].z = other_out[outIndex].x;
			other_out[outIndex].w = 255;
		}
	}
}

extern void LayerShower(int pikSize, uchar4 * other_out, int W, int H, Layer * showingLayer){

	float * dataPtr = showingLayer->Tensor;
	int layerSample = showingLayer->hTsample;
	int layerFeature = showingLayer->hTfeatureNum;
	int layerChanal = showingLayer->hTchanal;
	int layerHeigth = showingLayer->hTheigth;
	int layerWidth = showingLayer->hTwidth;
	int maxSample = W / (pikSize*layerSample);
	int maxFeature = (H / 2) / (pikSize*layerFeature);
	if (maxSample > layerSample) 
		maxSample = layerSample;
	if (maxFeature > layerFeature)
		maxFeature = layerFeature;
	if (maxSample >= layerSample && maxFeature >= layerFeature && layerChanal<=3){
		if (layerChanal == 1){
			
			int block = 8;
			dim3 blockSize(block, block,1);
			dim3 gridSize((W + blockSize.x - 1) / blockSize.x, (H + blockSize.y - 1) / blockSize.y,layerChanal);
		
			layerShowerCuda << <gridSize, blockSize >> > (other_out,pikSize, W, H, dataPtr, layerSample,
				layerFeature, layerChanal, layerHeigth, layerWidth,maxSample,maxFeature);
		}
	}

}