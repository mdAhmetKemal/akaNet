#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "ConnectActFunc.h"
#include <math.h>
#include <hip/hip_runtime.h>

__global__ void lineerDer(float * layer, int layerSize){

}
__global__ void ReluDer(float * layer, int layerSize){
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index < layerSize){
		if (layer[index] < 0){
			layer[index] = 0.0;
		}
		else{
			layer[index] = 1.0;
		}
	}
}
__global__ void LReluDer(float * layer, int layerSize, float parameter){
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index < layerSize){
		if (layer[index] < 0){
			layer[index] =  parameter;
		}
			else{
		layer[index] = 1.;
			}
	}
}
__global__ void EluDer(float * layer, int layerSize, float parameter){
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index < layerSize){
		if (layer[index] < 0){
			layer[index] = parameter+layer[index] ;
		}
			else{
		layer[index] = 1.;
		}
	}
}
__global__ void SigmDer(float * layer, int layerSize){
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index < layerSize){
		layer[index] = layer[index] * (1. - layer[index]);
	}
}
__global__ void TanhDer(float * layer, int layerSize){
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index < layerSize){
		layer[index] = 1.-(layer[index] * layer[index]);
	}
}
__global__ void StepDer(float * layer, int layerSize){
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index < layerSize){
		if (layer[index] > 0){
			layer[index] = 1.0;
		}
		else{
			layer[index] = 0.0;
		}
	}
}
__global__ void ArctanDer(float * layer, int layerSize){
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index < layerSize){
		layer[index] =1./ (layer[index] * layer[index]+1.);
	}
}
__global__ void SoftplusDer(float * layer, int layerSize){
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index < layerSize){
		layer[index] = 1./(1.+expf(-layer[index]));
	}
}
extern void DerivationFunction(_ConnectActType layerDer, float * layer, int layerSize, float parameter){
	int block = 512;
	dim3 BlockSize(block);
	dim3 GridSize(layerSize + BlockSize.x - 1 / BlockSize.x);
	if (layerDer == _NOACTV || layerDer == _LINEER){
		return;
	}
	if (layerDer == _RELU){
		ReluDer << <GridSize, BlockSize >> >(layer, layerSize);
	}
	if (layerDer == _ELU){
		EluDer << <GridSize, BlockSize >> >(layer, layerSize, parameter);
	}
	if (layerDer == _LRELU){
		LReluDer << <GridSize, BlockSize >> >(layer, layerSize, parameter);
	}
	if (layerDer == _SIGM){
		SigmDer << <GridSize, BlockSize >> >(layer, layerSize);
	}
	if (layerDer == _TANH){
		TanhDer << <GridSize, BlockSize >> >(layer, layerSize);
	}
	if (layerDer == _STEP){
		StepDer << <GridSize, BlockSize >> >(layer, layerSize);
	}
	if (layerDer == _ARCTAN){
		ArctanDer << <GridSize, BlockSize >> >(layer, layerSize);
	}
	if (layerDer == _SOFTPLUS){
		SoftplusDer << <GridSize, BlockSize >> >(layer, layerSize);
	}
}