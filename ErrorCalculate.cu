#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "ConnectCuda.h"
#include <hip/hip_runtime.h>

__global__ void ErrorCalculateDevice(const  float* __restrict__ outLayer,
	const  float* __restrict__ targetLayer,
	float * outParalelTensor,
	int totalPikselSize){
	int sampleNum = blockIdx.x*blockDim.x + threadIdx.x;
	float totalEr = 0;
	for (int p = 0; p < totalPikselSize; p++){
		totalEr += powf((outLayer[sampleNum*totalPikselSize + p] - targetLayer[sampleNum*totalPikselSize + p]), 2);
		printf("\nout:%.5f    target%.5f    in:%d", outLayer[sampleNum*totalPikselSize + p],
			targetLayer[sampleNum*totalPikselSize + p], p);
	}
//	totalEr = -sqrtf(totalEr);
	printf("\ntotalError %.4f ", totalEr);
	for (int p = 0; p < totalPikselSize; p++){
		outParalelTensor[sampleNum*totalPikselSize + p] = totalEr;
	}
	printf("\n");
}



void ErrorCalculateCu(float * outLayer, float* targetLayer, float * outParalelTensor, int sampleNum,int outFea,
	int outHe, int outWi, int outCha){
	int totalPikselSize =  outFea*outHe*outWi*outCha;

	dim3 BlockSize(1);
	dim3 GridSize(sampleNum );
	ErrorCalculateDevice << <GridSize, BlockSize >> >(outLayer, targetLayer, outParalelTensor, totalPikselSize); 

}