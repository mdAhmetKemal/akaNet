#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "ConnectCuda.h"





__global__ void conv2d_UpMom(
	float* kernelParalelTensor,
	int kernelFea,int kernelSize,float momentum
	){
	int kernelFeaInd = blockIdx.x*blockDim.x + threadIdx.x;
	int kernelIndex= blockIdx.y*blockDim.y + threadIdx.y;
	if (kernelFeaInd < kernelFea && kernelIndex < kernelSize){
		kernelParalelTensor[kernelFea*kernelIndex + kernelFeaInd] *= momentum;
		//printf("\n kernelIndex %d ", kernelFea*kernelIndex + kernelFeaInd);
	}
}
__global__ void conv2d_DelWeigth(
	const  float* __restrict__ inputWeLayerTensor, int inWeigthLaySamp,
	int inWeigthLayFea, int inWeigthLayHe, int inWeigthLayWi, int inWeigthLayCh,
	const  float* __restrict__ ErrorOutParalelTensor, int errOutFe, int errOutHe, int errOutWi, int errOutCh,
	float* kernelParalelTensor, int kernelFea, int kernelHeWeSc, int kernelDepth, int striX, int striY, int padX, int padY,
	int halfKernelSize, float learning){
	int forSamp = blockIdx.x*blockDim.x + threadIdx.x;
	int kerFea = blockIdx.y*blockDim.y + threadIdx.y;
	int outCentralIndex;
	int kernelIndex;
	int inHe, inWi, outHe, outWi;
	if (forSamp < inWeigthLaySamp && kerFea < kernelFea){
		for (int forFea = 0; forFea < inWeigthLayFea; forFea++){
			for (int keCha = 0; keCha < kernelDepth; keCha++){
				for (int keHe = 0; keHe < kernelHeWeSc; keHe++){
					for (int keWi = 0; keWi < kernelHeWeSc; keWi++){
						kernelIndex = kernelFea*(kernelHeWeSc*(kernelHeWeSc*(keCha)+keWi) + keHe) + kerFea;
						for (outHe = 0; outHe < errOutHe; outHe++){
							for (outWi = 0; outWi < errOutHe; outWi++){

								inHe = (outHe*striX) - padX - halfKernelSize + keHe;
								inWi = (outWi*striY) - padY - halfKernelSize + keWi;
								kernelParalelTensor[kernelIndex] +=
									inputWeLayerTensor[inWeigthLaySamp*(inWeigthLayFea*(inWeigthLayHe*(inWeigthLayWi*(keCha)+inWi) + inHe) + forFea) + forSamp] *
									ErrorOutParalelTensor[inWeigthLaySamp*(inWeigthLayFea*(kernelFea*(errOutHe*(errOutWi*(keCha)+outWi) + outHe) + kerFea) + forFea) + forSamp] * learning;
							}
						}
					}
				}
			}
		}
	}
}

__global__ void conv2d_UpDelta(
	const  float* __restrict__ kernelParalelTensor,
	float* kernelWeigthTensor,
	int kernelFea, int kernelSize){
	int forFea = blockIdx.x*blockDim.x + threadIdx.x;
	int kernelIndex = blockIdx.y*blockDim.y + threadIdx.y;
	if (forFea < kernelFea && kernelIndex<kernelSize){
		//printf("\n++ %f   %d", kernelWeigthTensor[kernelFea*kernelIndex + forFea], kernelFea*kernelIndex + forFea);
		kernelWeigthTensor[kernelFea*kernelIndex + forFea] -= kernelParalelTensor[kernelFea*kernelIndex + forFea];
		//printf("\n++ %f   %f", kernelWeigthTensor[kernelFea*kernelIndex + forFea], kernelParalelTensor[kernelFea*kernelIndex + forFea]);
		//kernelWeigthTensor[kernelFea*kernelIndex + forFea] = 0.5;
	}
}
__global__ void conv2d_showouterror(
	const  float* __restrict__ erroroutparalel
){
	int index = blockIdx.x*blockDim.x + threadIdx.x;


	//printf("\n+conUpd %.10f   %d", erroroutparalel[index], index);
		//kernelWeigthTensor[kernelFea*kernelIndex + forFea] -= kernelParalelTensor[kernelFea*kernelIndex + forFea];

}

__global__ void conv2d_DelBias(
	const  float* __restrict__ ErrorOutParalelTensor,
	int inWeigthLaySamp,int errOutFe, int errOutHe, int errOutWi, int errOutCh,
	float* biasParalelTensor, int kernelFea, float learning){
	int forSamp = blockIdx.x*blockDim.x + threadIdx.x;
	int forFea = blockIdx.y*blockDim.y + threadIdx.y;
	int forErrLaySize = blockIdx.z*blockDim.z + threadIdx.z;
	if (forSamp < inWeigthLaySamp &&forFea < kernelFea){
		biasParalelTensor[forFea] += ErrorOutParalelTensor[inWeigthLaySamp*(errOutFe*forErrLaySize + forFea) + forSamp] * learning;
	}
}

void conv2d_Update(float* inputWeLayerTensor, int inWeigthLaySamp,
	int inWeigthLayFea, int inWeigthLayHe, int inWeigthLayWi, int inWeigthLayCh,
	float * ErrorOutParalelTensor, int errOutFe, int errOutHe, int errOutWi, int errOutCh,
	float* kernelWeigthTensor, float* kernelParalelTensor, int kernelFea, int kernelHeWeSc, int kernelDepth, int striX, int striY, int padX, int padY,
	float *biasTensor, float* biasParalelTensor, float learning, float momentum){
/*	int inpfastSize = inWeigthLaySamp*inWeigthLayFea*inWeigthLayHe*inWeigthLayWi*inWeigthLayCh;
	int outFastSize = inWeigthLaySamp*totalOutLayFe*totalOutLayHe*totalOutLayWi*totalOutLayCh;
	int blokX = 16;
	int blokY = 16;
	int blokZ = 2;
	dim3 BlockSize(blokX, blokY, blokZ);
	dim3 GridSize((totalOutLayHe + BlockSize.x - 1) / BlockSize.x,
		(totalOutLayWi + BlockSize.y - 1) / BlockSize.y,
		(totalOutLayCh + BlockSize.z - 1) / BlockSize.z);
	hipMemset(totalOutLayerTensor, 0, outFastSize * sizeof(float));
	conv2d_FeedCu << <GridSize, BlockSize >> >(inputWeLayerTensor, inWeigthLaySamp,
		inWeigthLayFea, inWeigthLayHe, inWeigthLayWi, inWeigthLayCh,
		totalOutLayerTensor,
		totalOutLayFe, totalOutLayHe, totalOutLayWi, totalOutLayCh,
		kernelWeigthTensor,
		kernelFea, kernelHeWeSc, kernelDepth, striX, striY, padX, padY);
	dim3 blok2(16, 32);
	dim3 grid2((inWeigthLaySamp + BlockSize.x - 1) / BlockSize.x,
		(kernelFea + BlockSize.y - 1) / BlockSize.y);
	outFastSize = totalOutLayHe*totalOutLayWi*totalOutLayCh;
	conv2d_FeedBias << <grid2, blok2 >> >(inWeigthLaySamp, kernelFea, totalOutLayerTensor, outFastSize, biasTensor);
	*/
	int errorLayerSize = inWeigthLaySamp*errOutFe*errOutHe*errOutWi* errOutCh;
	int kernelSize = kernelHeWeSc*kernelHeWeSc*kernelDepth;
	int halfKernelSize = (kernelHeWeSc - 1) / 2;
	int blokA = 16;
	int blokB = 16;
	dim3 grido(errorLayerSize);
	conv2d_showouterror << <grido, 1 >> >(ErrorOutParalelTensor);
	dim3 BlockSize(blokA,blokB,1);
	dim3 GridSize((kernelFea + BlockSize.x - 1) / BlockSize.x, (kernelSize + BlockSize.y- 1) / BlockSize.y);
	conv2d_UpMom <<< GridSize, BlockSize >> > (kernelParalelTensor,kernelFea,kernelSize,momentum);
	
	GridSize.x = (kernelFea + BlockSize.x - 1) / BlockSize.x;
	GridSize.y = 1; BlockSize.y = 1;
	conv2d_UpMom << < GridSize, BlockSize >> > (biasParalelTensor, kernelFea, 1, momentum);
	//******buraya kadar do�ruuuuu
	//hipMemset(kernelParalelTensor, 0, kernelSize * sizeof(float));..s
	GridSize.x = (inWeigthLaySamp + BlockSize.x - 1) / BlockSize.x;
	BlockSize.y = blokB;
	GridSize.y = (kernelFea + BlockSize.y - 1) / BlockSize.y;
	conv2d_DelWeigth << <GridSize, BlockSize >> >(inputWeLayerTensor, inWeigthLaySamp, inWeigthLayFea,inWeigthLayHe,inWeigthLayWi,
		inWeigthLayCh, ErrorOutParalelTensor, errOutFe, errOutHe, errOutWi, errOutCh, kernelParalelTensor, kernelFea, kernelHeWeSc,
		kernelDepth, striX, striY, padX, padY, halfKernelSize,learning);
	GridSize.z = errorLayerSize ;
	//hipMemset(biasParalelTensor, 0, kernelFea * sizeof(float));
	/*conv2d_DelBias << <GridSize, BlockSize >> >(ErrorOutParalelTensor, inWeigthLaySamp,errOutFe, errOutHe, errOutWi, errOutCh,
		biasParalelTensor,
		kernelFea,
		learning); */
	dim3 BlockSize2(16, 16);
	dim3 GridSize2((kernelFea + BlockSize.x - 1) / BlockSize.x, (kernelSize + BlockSize.y - 1) / BlockSize.y);
	conv2d_UpDelta << <GridSize2, BlockSize2 >> >(kernelParalelTensor,kernelWeigthTensor, kernelFea, kernelSize);
	GridSize2.y = 1;
	//conv2d_UpDelta << <GridSize2, BlockSize2 >> >(biasTensor, biasParalelTensor, kernelFea, 1);

};